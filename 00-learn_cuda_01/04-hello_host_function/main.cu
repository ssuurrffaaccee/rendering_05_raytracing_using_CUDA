#include <cstdio>
#include <hip/hip_runtime.h>

__device__ void say_hello() {
    printf("Hello, world from GPU!\n");
}

__host__ void say_hello_host() {
    printf("Hello, world from CPU!\n");
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();//stop, then wait gpu to finish
    say_hello_host();
    return 0;
}
